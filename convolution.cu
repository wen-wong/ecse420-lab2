#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "gputimer.h"

#include <hip/hip_runtime.h>
#include <>

#include "lodepng.h"
#include "wm.h"

__global__ void convolution(unsigned char *input, unsigned char *output, float *wm, int width, int height) {
    int item_count = (width - 2) * (height - 2);
    int op_per_thread = item_count / (blockDim.x);
    if (item_count % (blockDim.x)) {
        op_per_thread++;
    }

    for(int op = 0; op < op_per_thread; op++) {
        int index = threadIdx.x * op_per_thread + op;
        // if (threadIdx.x == 0) {
        //     printf("op: %d, index: %d\n", op, index);
        // } else {
        //     break;
        // }

        if (index >= item_count) {
            return;
        }
        // printf("index: %d\n", index);
        int i = index / (width - 2);
        int j = index % (width - 2);

        if (!(i >= 1 && i <= (height) - 1) || !(j >= 1 && j <= (width) - 1)) {
            continue;
        }

        float r = 0, g = 0, b = 0, a = 0;

        for(int ii = 0; ii <= 2; ii++) {
            for(int jj = 0; jj <= 2; jj++) {
                // printf("i: %d, j: %d, ii: %d, jj: %d\n", i, j, ii, jj);
                for(int k = 0; k <= 3; k++) {
                    // output[index * 4 + k] += input[(i + ii - 1) * width * 4 + (j + jj - 1) * 4 + k] * wm[ii * 3 + jj];
                    // if (k == 3) {
                    //     output[index * 4 + k] = 255;
                    // } else {
                    //     output[index * 4 + k] += input[index * 4 + k];
                    // }
                    // output[index * 4 + k] 
                    r += input[(i + ii - 1) * width * 4 + (j + jj - 1) * 4] * wm[ii * 3 + jj];
                    g += input[(i + ii - 1) * width * 4 + (j + jj - 1) * 4 + 1] * wm[ii * 3 + jj];
                    b += input[(i + ii - 1) * width * 4 + (j + jj - 1) * 4 + 2] * wm[ii * 3 + jj];
                    a += input[(i + ii - 1) * width * 4 + (j + jj - 1) * 4 + 3] * wm[ii * 3 + jj];
                    // if (index == 10000) {
                    //     printf("[%d][%d] k: %d\n", (i + ii - 1) + (i * width), (j + jj - 1), k);
                    // }
                }
            }
        }

        output[index * 4] = r;
        output[index * 4 + 1] = g;
        output[index * 4 + 2] = b;
        output[index * 4 + 3] = a;
    }
}

int main(int argc, char *argv[]) {
    char* input_filename = argv[1];
    char* output_filename = argv[2];
    int num_of_threads = atoi(argv[3]);

    unsigned width, height;

    unsigned error;
    unsigned char *temp_image;
    unsigned char *temp_output;
    float *wm;
    unsigned char *input_image;
    unsigned char *output_image;

    error = lodepng_decode32_file(&temp_image, &width, &height, input_filename);
    if (error) {
        printf("error %u: %s", error, lodepng_error_text(error));
    }

    temp_output = (unsigned char *) malloc((width - 2) * (height - 2) * 4 * sizeof(unsigned char));

    // Allocate memory for input and output images
    hipMalloc((void **) &input_image, width * height * 4 * sizeof(unsigned char));
    hipMalloc((void **) &output_image, (width - 2) * (height - 2) * 4 * sizeof(unsigned char));
    hipMalloc((void **) &wm, 3 * 3 * sizeof(float));


    // Copy input image to device
    hipMemcpy(input_image, temp_image, width * height * 4 * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(wm, w, 3 * 3 * sizeof(float), hipMemcpyHostToDevice);

    GpuTimer timer;
    timer.Start();

    convolution<<<1, num_of_threads>>>(input_image, output_image, wm, (int) width, (int) height);

    hipDeviceSynchronize();

    timer.Stop();
    printf("*** Time Elapsed: %g ms ***\n", timer.Elapsed());

    printf("error: %s\n", hipGetErrorString(hipGetLastError()));

    // Copy output image to host
    hipMemcpy(temp_output, output_image, (width - 2) * (height - 2) * 4 * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Save output image
    lodepng_encode32_file(output_filename, temp_output, width - 2, height - 2);
    
    // Free memory
    free(temp_image);
    free(temp_output);
    hipFree(input_image);
    hipFree(output_image);
    
    return 0;
}