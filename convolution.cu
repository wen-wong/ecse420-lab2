#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "gputimer.h"

#include <hip/hip_runtime.h>
#include <>

#include "lodepng.h"

__global__ void convolution(unsigned char *input, unsigned char *output, int width, int height) {
    int item_count = (width - 2) * (height - 2);

    int op_per_thread = item_count / (blockDim.x);
    if (item_count % (blockDim.x)) {
        op_per_thread++;
        
    }
    printf("op per thread: %d", op_per_thread);

    for(int op = 0; op < op_per_thread; op++) {
        int index = threadIdx.x + (blockDim.x) * op;

        if (index < item_count) {
        // Implement convolution
        printf("index: %d\n", index);
            int i = index / (width - 2);
            int j = index % (width - 2);

            // printf("%d %d\n", i, j);

            // for(int ii = 0; ii <= 2; ii++) {
            //     for(int jj = 0; jj <= 2; jj++) {

            //     }
            // }

        }   
    }
}

int main(int argc, char **argv) {
    char* input_png = argv[1];
    char* output_png = argv[2];
    int num_of_threads = atoi(argv[3]);

    unsigned width, height;


    unsigned error;
    unsigned char *temp_image;
    unsigned char *temp_output;
    unsigned char *input_image;
    unsigned char *output_image;

    error = lodepng_decode32_file(&temp_image, &width, &height, input_png);
    if (error) {
        printf("error %u: %s", error, lodepng_error_text(error));
    }

    temp_output = (unsigned char *) malloc((width - 2) * (height - 2) * 4 * sizeof(unsigned char));

    // Allocate memory for input and output images
    hipMalloc((void **) &input_image, width * height * 4 * sizeof(unsigned char));
    hipMalloc((void **) &output_image, (width - 2) * (height - 2) * 4 * sizeof(unsigned char));


    // Copy input image to device
    hipMemcpy(input_image, temp_image, width * height * 4 * sizeof(unsigned char), hipMemcpyHostToDevice);

    GpuTimer timer;
    timer.Start();

    convolution<<<1, num_of_threads>>>(input_image, output_image, width, height);

    hipDeviceSynchronize();

    timer.Stop();
    printf("*** Time Elapsed: %g ms ***\n", timer.Elapsed());

    printf("error: %s\n", hipGetErrorString(hipGetLastError()));

    // Copy output image to host
    hipMemcpy(temp_output, output_image, (width - 2) * (height - 2) * 4 * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Save output image
    lodepng_encode32_file(output_png, temp_output, width - 2, height - 2);
    
    // Free memory
    free(temp_image);
    free(temp_output);
    hipFree(input_image);
    hipFree(output_image);
    
    return 0;
}