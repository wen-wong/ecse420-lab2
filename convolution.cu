#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "gputimer.h"

#include <hip/hip_runtime.h>
#include <>

#include "lodepng.h"
#include "wm.h"

__global__ void convolution(unsigned char *input, unsigned char *output, float *wm, int width, int height) {
    int item_count = (width - 2) * (height - 2);
    int op_per_thread = item_count / (blockDim.x);
    if (item_count % (blockDim.x)) {
        op_per_thread++;
    }

    for(int op = 0; op < op_per_thread; op++) {
        int index = threadIdx.x * op_per_thread + op;
        // int index = op * (blockDim.x) + threadIdx.x;
        if (index >= item_count) {
            return;
        }
        // printf("index: %d\n", index);
        int i = index / (width - 2);
        int j = index % (width - 2);

        if (!(1 <= i && i <= (width) - 1) && !(1 <= j && j <= (height) - 1)) {
            return;
        }

        for(unsigned long ii = 0; ii <= 2; ii++) {
            for(unsigned long jj = 0; jj <= 2; jj++) {
                for(int k = 0; k <= 3; k++) {
                    // output[index * 4 + k] += input[(i + ii - 1) * width * 4 + (j + jj - 1) * 4 + k] * wm[ii * 3 + jj];
                    // if (k == 3) {
                    //     output[index * 4 + k] = 255;
                    // } else {
                    //     output[index * 4 + k] += input[index * 4 + k];
                    // }
                    output[index * 4 + k] += input[(i + ii - 1) * width * 4 + (j + jj - 1) * 4 + k] * wm[ii * 3 + jj];
                }
            }
        }
    }
}

int main(int argc, char *argv[]) {
    char* input_filename = argv[1];
    char* output_filename = argv[2];
    int num_of_threads = atoi(argv[3]);

    unsigned width, height;

    unsigned error;
    unsigned char *temp_image;
    unsigned char *temp_output;
    float *wm;
    unsigned char *input_image;
    unsigned char *output_image;

    error = lodepng_decode32_file(&temp_image, &width, &height, input_filename);
    if (error) {
        printf("error %u: %s", error, lodepng_error_text(error));
    }

    temp_output = (unsigned char *) malloc((width - 2) * (height - 2) * 4 * sizeof(unsigned char));

    // Allocate memory for input and output images
    hipMalloc((void **) &input_image, width * height * 4 * sizeof(unsigned char));
    hipMalloc((void **) &output_image, (width - 2) * (height - 2) * 4 * sizeof(unsigned char));
    hipMalloc((void **) &wm, 3 * 3 * sizeof(float));


    // Copy input image to device
    hipMemcpy(input_image, temp_image, width * height * 4 * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(wm, w, 3 * 3 * sizeof(float), hipMemcpyHostToDevice);

    GpuTimer timer;
    timer.Start();

    convolution<<<1, num_of_threads>>>(input_image, output_image, wm, (int) width, (int) height);

    hipDeviceSynchronize();

    timer.Stop();
    printf("*** Time Elapsed: %g ms ***\n", timer.Elapsed());

    printf("error: %s\n", hipGetErrorString(hipGetLastError()));

    // Copy output image to host
    hipMemcpy(temp_output, output_image, (width - 2) * (height - 2) * 4 * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Save output image
    lodepng_encode32_file(output_filename, temp_output, width - 2, height - 2);
    
    // Free memory
    free(temp_image);
    free(temp_output);
    hipFree(input_image);
    hipFree(output_image);
    
    return 0;
}